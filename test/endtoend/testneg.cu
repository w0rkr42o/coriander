
#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void getValue(float *indata) {
    indata[0] = 0.0f - indata[0];
}

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipMalloc(&deviceFloats1, N * sizeof(float));

    hostFloats1[0] = 123.456f;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats1)),
        hostFloats1,
        N * sizeof(float),
        stream
    );

    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>((float *)deviceFloats1);

    hipMemcpyDtoHAsync(hostFloats1, deviceFloats1, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;

    assert(hostFloats1[0] == -123.456f);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);
    hipStreamDestroy(stream);

    return 0;
}
